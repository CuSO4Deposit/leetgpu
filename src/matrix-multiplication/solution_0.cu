#include "hip/hip_runtime.h"
#include "solve.h"
#include <hip/hip_runtime.h>

__global__ void matrix_multiplication_kernel(const float* A, const float* B, float* C, int M, int N, int K) {
    size_t x = blockDim.x * blockIdx.x + threadIdx.x;
    size_t y = blockDim.y * blockIdx.y + threadIdx.y;
    size_t idx = y * K + x;
    if (x < K && y < M) {
        float sum = 0;
        for (size_t k = 0; k < N; k++) {
            sum += A[y * N + k] * B[k * K + x];
        }
        C[idx] = sum;
    };
}

// A, B, C are device pointers (i.e. pointers to memory on the GPU)
void solve(const float* A, const float* B, float* C, int M, int N, int K) {
    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((K + threadsPerBlock.x - 1) / threadsPerBlock.x,
                       (M + threadsPerBlock.y - 1) / threadsPerBlock.y);
    
    matrix_multiplication_kernel<<<blocksPerGrid, threadsPerBlock>>>(A, B, C, M, N, K);
    hipDeviceSynchronize();
}
